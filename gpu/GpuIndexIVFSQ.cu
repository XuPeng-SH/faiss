#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include "GpuIndexIVFSQ.h"
#include "../IndexFlat.h"
#include "../IndexIVFFlat.h"
#include "GpuIndexFlat.h"
#include "GpuResources.h"
#include "impl/IVFSQ.cuh"
#include "utils/CopyUtils.cuh"
#include "utils/DeviceUtils.h"

#include <limits>
#include <memory>

namespace faiss { namespace gpu {

GpuIndexIVFSQ::GpuIndexIVFSQ(GpuResources* resources,
        const CpuIndexT* index,
        ConfigT config) :
    BaseT(resources, index->d, index->metric_type, index->nlist, config),
    config_(config),
    reserveMemoryVecs_(0),
    index_(nullptr) {
    copyFrom(index);
}

GpuIndexIVFSQ::GpuIndexIVFSQ(GpuResources* resources,
                                 int dims,
                                 int nlist,
                                 MetricT metric,
                                 ConfigT config) :
    BaseT(resources, dims, metric, nlist, config),
    config_(config),
    reserveMemoryVecs_(0),
    index_(nullptr) {
  this->is_trained = false;
}

GpuIndexIVFSQ::~GpuIndexIVFSQ() {
  delete index_;
}

void
GpuIndexIVFSQ::reserveMemory(size_t numVecs) {
  reserveMemoryVecs_ = numVecs;
  if (index_) {
    index_->reserveMemory(numVecs);
  }
}

void
GpuIndexIVFSQ::copyFrom(const CpuIndexT* index) {
  DeviceScope scope(device_);

  BaseT::copyFrom(index);

  // Clear out our old data
  delete index_;
  index_ = nullptr;

  // The other index might not be trained
  if (!index->is_trained) {
    return;
  }

  // Otherwise, we can populate ourselves from the other index
  this->is_trained = true;

  float vmin = index->sq.trained[0];
  float vdiff = index->sq.trained[1];


  index_ = new ImplT(resources_,
          quantizer_->getGpuData(),
          index->sq.code_size,
          index->metric_type == faiss::METRIC_L2,
          config_.indicesOptions,
          memorySpace_,
          vmin,
          vdiff);

  InvertedLists *ivf_lists = index->invlists;

  for (size_t i = 0; i < ivf_lists->nlist; ++i) {
    auto numVecs = ivf_lists->list_size(i);

    FAISS_THROW_IF_NOT_FMT(numVecs <=
                       (size_t) std::numeric_limits<int>::max(),
                       "GPU inverted list can only support "
                       "%zu entries; %zu found",
                       (size_t) std::numeric_limits<int>::max(),
                       numVecs);

    index_->addCodeVectorsFromCpu(
             i, ivf_lists->get_codes(i),
             ivf_lists->get_ids(i), numVecs);
  }
}

void
GpuIndexIVFSQ::dump() {
    for (auto i=0; i<index_->getNumLists(); ++i) {
        std::cout << "Size Of Buckets[" << i <<  "] = " << index_->getListLength(i) << std::endl;
    }
    for (auto i=0; i<index_->getNumLists(); ++i) {
        std::cout << "GpuIndice[" << i << "] = ";
        auto indices = index_->getListIndices(i);
        for (auto& id : indices) {
            std::cout << id << " | ";
        }
        std::cout << std::endl;
    }

    std::cout << "Trained data size = "  << index_->getTrainedData()->capacity() << std::endl;
}

void
GpuIndexIVFSQ::copyTo(CpuIndexT* index) const {
  FAISS_THROW_MSG("GpuIndexIVFSQ train not supported");
}

size_t
GpuIndexIVFSQ::reclaimMemory() {
  if (index_) {
    DeviceScope scope(device_);

    return index_->reclaimMemory();
  }

  return 0;
}

void
GpuIndexIVFSQ::reset() {
  if (index_) {
    DeviceScope scope(device_);

    index_->reset();
    this->ntotal = 0;
  } else {
    FAISS_ASSERT(this->ntotal == 0);
  }
}

void
GpuIndexIVFSQ::train(Index::idx_t n, const float* x) {
  FAISS_THROW_MSG("GpuIndexIVFSQ train not supported");
}

void
GpuIndexIVFSQ::addImpl_(int n,
                          const float* x,
                          const Index::idx_t* xids) {
  FAISS_THROW_MSG("GpuIndexIVFSQ addImpl_ not supported");
}

void
GpuIndexIVFSQ::searchImpl_(int n,
                             const float* x,
                             int k,
                             float* distances,
                             Index::idx_t* labels) const {
  // Device is already set in GpuIndex::search
  FAISS_ASSERT(index_);
  FAISS_ASSERT(n > 0);

  // Data is already resident on the GPU
  Tensor<float, 2, true> queries(const_cast<float*>(x), {n, (int) this->d});
  Tensor<float, 2, true> outDistances(distances, {n, k});

  static_assert(sizeof(long) == sizeof(Index::idx_t), "size mismatch");
  Tensor<long, 2, true> outLabels(const_cast<long*>(labels), {n, k});

  index_->query(queries, nprobe_, k, outDistances, outLabels);
}

} } // namespace
