#include "hip/hip_runtime.h"

#include "IVFSQ.cuh"
#include "../GpuResources.h"
#include "FlatIndex.cuh"
#include "InvertedListAppend.cuh"
#include "IVFSQScan.cuh"
#include "RemapIndices.h"
#include "../utils/CopyUtils.cuh"
#include "../utils/DeviceDefs.cuh"
#include "../utils/DeviceUtils.h"

#include "../utils/HostTensor.cuh"
#include "../utils/Transpose.cuh"
#include <limits>
#include <thrust/host_vector.h>
#include <unordered_map>
 
namespace faiss { namespace gpu {

IVFSQ::IVFSQ(GpuResources* resources,
                CoarseQuantizerT* coarse_quantizer,
                int code_size,
                bool l2Distance,
                IndicesOptions indicesOptions,
                MemorySpace space,
                float vdiff,
                float vmin) :
    IVFBase(resources,
            coarse_quantizer,
            code_size * coarse_quantizer->getDim(),
            indicesOptions,
            space),
    l2Distance_(l2Distance),
    gpu_scalar_quantizer_(vdiff, vmin) {
}
 
IVFSQ::~IVFSQ() {
}

void
IVFSQ::addCodeVectorsFromCpu(int listId,
        const VecT* vecs, const long* indices, size_t numVecs) {
    FAISS_ASSERT(listId < deviceListData_.size());
    auto stream = resources_->getDefaultStreamCurrentDevice();

    if (numVecs == 0) {
        return;
    }

    size_t lengthInBytes = numVecs * bytesPerVector_;

    auto& listData = deviceListData_[listId];
    auto prevData = listData->data();

    FAISS_ASSERT(listData->size() + lengthInBytes <= (size_t)std::numeric_limits<int>::max());

    listData->append((unsigned char*) vecs,
            lengthInBytes,
            stream,
            true);

    addIndicesFromCpu_(listId, indices, numVecs);

    if (prevData != listData->data()) {
        deviceListDataPointers_[listId] = listData->data();
    }

    int listLength = listData->size() / bytesPerVector_;
    deviceListLengths_[listId] = listLength;

    maxListLength_ = std::max(maxListLength_, listLength);

    if (stream != 0) {
        streamWait({stream}, {0});
    }
}

void
IVFSQ::addTrainedDataFromCpu(const uint8_t* trained,
                            size_t numData) {
    addTrainedDataFromCpu_(trained, numData);
}

std::vector<uint8_t>
IVFSQ::getListVectors(int listId) const {
  FAISS_ASSERT(listId < deviceListData_.size());
  auto& encVecs = *deviceListData_[listId];

  auto stream = resources_->getDefaultStreamCurrentDevice();

  size_t num = encVecs.size() / sizeof(uint8_t);

  Tensor<uint8_t, 1, true> dev((uint8_t*) encVecs.data(), {(int) num});

  std::vector<uint8_t> out(num);
  HostTensor<uint8_t, 1, true> host(out.data(), {(int) num});
  host.copyFrom(dev, stream);

  return out;
}

 void
 IVFSQ::query(Tensor<float, 2, true>& queries,
                int nprobe,
                int k,
                Tensor<float, 2, true>& outDistances,
                Tensor<long, 2, true>& outIndices) {
   auto& mem = resources_->getMemoryManagerCurrentDevice();
   auto stream = resources_->getDefaultStreamCurrentDevice();
 
   // These are caught at a higher level
   FAISS_ASSERT(nprobe <= GPU_MAX_SELECTION_K);
   FAISS_ASSERT(k <= GPU_MAX_SELECTION_K);
   nprobe = std::min(nprobe, quantizer_->getSize());
 
   FAISS_ASSERT(queries.getSize(1) == dim_);
 
   FAISS_ASSERT(outDistances.getSize(0) == queries.getSize(0));
   FAISS_ASSERT(outIndices.getSize(0) == queries.getSize(0));
 
   // Reserve space for the quantized information
   DeviceTensor<float, 2, true>
     coarseDistances(mem, {queries.getSize(0), nprobe}, stream);
   DeviceTensor<int, 2, true>
     coarseIndices(mem, {queries.getSize(0), nprobe}, stream);
 
   // Find the `nprobe` closest lists; we can use int indices both
   // internally and externally
   quantizer_->query(queries,
                     nprobe,
                     coarseDistances,
                     coarseIndices,
                     false);
 
   runIVFScalarQuantizerScan(queries,
                  coarseIndices,
                  deviceListDataPointers_,
                  deviceListIndexPointers_,
                  indicesOptions_,
                  deviceListLengths_,
                  maxListLength_,
                  k,
                  l2Distance_,
                  outDistances,
                  outIndices,
                  resources_,
                  gpu_scalar_quantizer_);
 
   // If the GPU isn't storing indices (they are on the CPU side), we
   // need to perform the re-mapping here
   // FIXME: we might ultimately be calling this function with inputs
   // from the CPU, these are unnecessary copies
   if (indicesOptions_ == INDICES_CPU) {
     HostTensor<long, 2, true> hostOutIndices(outIndices, stream);
 
     ivfOffsetToUserIndex(hostOutIndices.data(),
                          numLists_,
                          hostOutIndices.getSize(0),
                          hostOutIndices.getSize(1),
                          listOffsetToUserIndex_);
 
     // Copy back to GPU, since the input to this function is on the
     // GPU
     outIndices.copyFrom(hostOutIndices, stream);
   }
 }
 
 } } // namespace
 